#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THC.h>

#include <stdio.h>
#include <math.h>
#include <algorithm>

#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N)
{
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

namespace cvpods{

template <typename scalar_t>
__global__ void CropSplitGtKernelForward(
    const int count,
    const scalar_t *bottom_data,
    const scalar_t *bottom_rois,
    const int height,
    const int width,
    const int num_cell,
    const int num_box,
    scalar_t *top_data)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    // The output is in order (n, ctop, ph, pw)
    //int pw = index % width;
    //int ph = (index / width) % height;
    //int n = index / width / height;
    int n = index % num_box;
    int pw = (index / num_box) % width;
    int ph = index / num_box / width;
    // [start, end) interval for spatial sampling
    const scalar_t *offset_bottom_rois = bottom_rois + n * 4;
    scalar_t roi_x1 = offset_bottom_rois[0];
    scalar_t roi_y1 = offset_bottom_rois[1];
    scalar_t roi_x2 = offset_bottom_rois[2];
    scalar_t roi_y2 = offset_bottom_rois[3];

    if((pw >= roi_x1) & (ph >= roi_y1) & (pw < roi_x2) & (ph < roi_y2)){
        top_data[index] = bottom_data[index];
    }
   }
}

void CropSplitGtForward(const at::Tensor data,
                        const at::Tensor bbox,
                        at::Tensor out,
                        const int height,
                        const int width,
                        const int num_cell,
                        const int num_bbox)
{
  const int count = num_bbox * height * width;
  //printf("aa, %d ",count);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data.type(), "CropSplitGtForward", ([&] {
        const scalar_t *bottom_data = data.data<scalar_t>();
        const scalar_t *bottom_rois = bbox.data<scalar_t>();
        scalar_t *top_data = out.data<scalar_t>();

        CropSplitGtKernelForward<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, bottom_data, bottom_rois, height, width, num_cell, num_bbox, top_data);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in CropSplitGtForward: %s\n", hipGetErrorString(err));
  }
}


template <typename scalar_t>
__global__ void CropSplitGtKernelBack(
    const int count,
    const scalar_t *top_diff,
    const scalar_t *bottom_rois,
    const int height,
    const int width,
    const int num_cell,
    const int num_box,
    scalar_t *bottom_diff)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    // The output is in order (n, ctop, ph, pw)
    //int pw = index % width;
    //int ph = (index / width) % height;
    //int n = index / width / height;
    int n = index % num_box;
    int pw = (index / num_box) % width;
    int ph = index / num_box / width;
    // [start, end) interval for spatial sampling
    const scalar_t *offset_bottom_rois = bottom_rois + n * 4;
    scalar_t roi_x1 = offset_bottom_rois[0];
    scalar_t roi_y1 = offset_bottom_rois[1];
    scalar_t roi_x2 = offset_bottom_rois[2];
    scalar_t roi_y2 = offset_bottom_rois[3];
    if((pw >= roi_x1) & (ph >= roi_y1) & (pw < roi_x2) & (ph < roi_y2)){
        atomicAdd(bottom_diff+index, top_diff[index]);
    }
   }
}


void CropSplitGtBack(const at::Tensor top_grad,
                    const at::Tensor bbox,
                    at::Tensor bottom_grad,
                    const int height,
                    const int width,
                    const int num_cell,
                    const int num_bbox)
{
  const int count = num_bbox * height * width;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "CropSplitGtBack", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *bottom_rois = bbox.data<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();

        CropSplitGtKernelBack<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_rois, height, width, num_cell, num_bbox, bottom_diff);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in CropSplitGtBack: %s\n", hipGetErrorString(err));
  }

}

}